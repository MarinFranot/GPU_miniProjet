#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <chrono>
#include <iostream>
#include <fstream>


bool checkCuda(int* out_cpu, int* out_gpu, int N);

#define CHK(code) \
do { \
    if ((code) != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s %s %i\n", \
                        hipGetErrorString((code)), __FILE__, __LINE__); \
        goto Error; \
    } \
} while (0)



__global__ void addKernel(int* c, const int* a, const int* b, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N)
        return;

    c[i] = a[i] + b[i];
}

__global__ void addKernel2(int* c, const int* a, const int* b, int N, int J)
{
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * J;
    if (i >= N)
        return;
    for (int j = 0; j < J; j++)
        c[i + j] = a[i + j] + b[i + j];
}

__global__ void addKernel3(int* c, const int* a, const int* b, int N, int K)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N)
        return;

    c[i] = a[i];
    for (int k = 0; k < K; k++)
        c[i] += b[i];
}

__global__ void mulKernel(int* c, const int* a, const int* b, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N)
        return;

    c[i] = a[i] * b[i];
}

__global__ void mulKernel2(int* c, const int* a, const int* b, int N, int J)
{
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * J;
    if (i >= N)
        return;
    for (int j = 0; j < J; j++)
        c[i + j] = a[i + j] * b[i + j];
}

__global__ void mulKernel3(int* c, const int* a, const int* b, int N, int K)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N)
        return;

    c[i] = a[i];
    for (int k = 0; k < K; k++)
        c[i] *= b[i];
}

#define N_THREAD 128
#define Ntest 10


int main()
{
    int MODE = 3; // MODE: {1: simple addition, 2:addition avec du J, 3: addition avec du K}
                  // Si MODE = 1, alors les calcules font aussi être sur le CPU
                  // Si MODE > 1, le CPU ne va pas être utilisé
    int OP = 2;   // Op: Opérator: {1: addition, 2:multiplication}
    int J = 1;
    int K = 200;

    std::string name = "result/K_mul.txt";
    std::ofstream fichier(name, std::ios::out | std::ios::trunc);
    // fichier << "taille|temps_cpu|temps_gpu|memoryThroughput(GB/s)|computationThroughput(GOPS/s)" << std::endl;
    fichier << "taille|K|temps_gpu|memoryThroughput(GB/s)|computationThroughput(GOPS/s)|computeIntensity(OPS/Byte)" << std::endl;
    fichier << "Operator = " << OP << " | ";
    fichier << "Ntest = " << Ntest << " | ";
    fichier << "Mode = " << MODE << " | ";
    fichier << "N_thread = " << N_THREAD << std::endl;

    for (int puissance = 6; puissance < 7; puissance++) {
    
        // std::cout << puissance << std::endl;
        
        for (int k = 1; k < K; k++) {

            const int arraySize = pow(10, puissance); // On fait une taille de 10^(puissance)
            std::cout << k << std::endl;

            float temps_cpu = 0;
            float temps_gpu = 0;

            for (int nbtest = 0; nbtest < Ntest; nbtest++) {

                int* h_a = (int*)malloc(arraySize * sizeof(int));
                int* h_b = (int*)malloc(arraySize * sizeof(int));
                int* h_c = (int*)malloc(arraySize * sizeof(int));


                for (int i = 0; i < arraySize; i++) {
                    h_a[i] = 1 + i % 100;
                    h_b[i] = 1 + (arraySize - i) % 100;
                }
                
                //Computation on CPU
                if (MODE == 1) {

                    int* h_cpu_result = (int*)malloc(arraySize * sizeof(int));

                    std::chrono::steady_clock::time_point start_cpu = std::chrono::high_resolution_clock::now();

                    for (int i = 0; i < arraySize; i++) {
                        h_cpu_result[i] = h_a[i] * h_b[i];
                    }
                    std::chrono::steady_clock::time_point stop_cpu = std::chrono::high_resolution_clock::now();
                    auto cpu_runtime_us = std::chrono::duration_cast<std::chrono::microseconds>(stop_cpu - start_cpu).count();

                    temps_cpu += cpu_runtime_us;
                }


                //2. Do the computation on GPU and time it

                // Define the variable we need 
                int* dev_a = 0;
                int* dev_b = 0;
                int* dev_c = 0;

                hipError_t cudaStatus;
                hipEvent_t start_gpu, stop_gpu; //cudaEvent are used to time the kernel
                hipEventCreate(&start_gpu);
                hipEventCreate(&stop_gpu);

                // Define the size of the grid (block_size = #blocks in the grid)
                //and the size of a block (thread_size = #threads in a block)
                //TODO 1) Change how block_size and thread_size are defined to work with bigger vectors 
                dim3 block_size((arraySize + (N_THREAD - 1)) / N_THREAD);
                dim3 thread_size(N_THREAD);

                // Choose which GPU to run on, change this on a multi-GPU system.
                CHK(hipSetDevice(0));


                // Allocate GPU buffers for three vectors (two input, one output)    .
                CHK(hipMalloc((void**)&dev_c, arraySize * sizeof(int)));
                CHK(hipMalloc((void**)&dev_a, arraySize * sizeof(int)));
                CHK(hipMalloc((void**)&dev_b, arraySize * sizeof(int)));

                // Copy input vectors from host memory to GPU buffers.
                CHK(hipMemcpy(dev_a, h_a, arraySize * sizeof(int), hipMemcpyHostToDevice));
                CHK(hipMemcpy(dev_b, h_b, arraySize * sizeof(int), hipMemcpyHostToDevice));;

                // Launch a kernel on the GPU with one thread for each element and time the kernel
                hipEventRecord(start_gpu);

                if (OP == 1) {  // Addition
                    if (MODE == 1)
                        addKernel << <block_size, thread_size >> > (dev_c, dev_a, dev_b, arraySize);
                    if (MODE == 2)
                        addKernel2 << <block_size, thread_size >> > (dev_c, dev_a, dev_b, arraySize, J);
                    if (MODE == 3)
                        addKernel3 << <block_size, thread_size >> > (dev_c, dev_a, dev_b, arraySize, K);
                }
                if (OP == 2) {  // Multiplication
                    if (MODE == 1)
                        mulKernel << <block_size, thread_size >> > (dev_c, dev_a, dev_b, arraySize);
                    if (MODE == 2)
                        mulKernel2 << <block_size, thread_size >> > (dev_c, dev_a, dev_b, arraySize, J);
                    if (MODE == 3)
                        mulKernel3 << <block_size, thread_size >> > (dev_c, dev_a, dev_b, arraySize, k);
                }

                hipEventRecord(stop_gpu);


                // Check for any errors launching the kernel
                CHK(hipGetLastError());


                // hipDeviceSynchronize waits for the kernel to finish, and returns
                // any errors encountered during the launch.
                CHK(hipDeviceSynchronize());


                // Copy output vector from GPU buffer to host memory.
                CHK(hipMemcpy(h_c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost));


                // Make sure the stop_gpu event is recorded before doing the time computation
                hipEventSynchronize(stop_gpu);
                float gpu_runtime_ms;
                hipEventElapsedTime(&gpu_runtime_ms, start_gpu, stop_gpu);


                temps_gpu += gpu_runtime_ms;

                /*
                if (!checkCuda(h_cpu_result, h_c, arraySize)) {
                    printf("ERROR GPU results are not corrrrrect !!!\n");
                }
                */
                

            Error:
                hipFree(dev_c);
                hipFree(dev_a);
                hipFree(dev_b);

                delete[] h_a, h_b, h_c;

                // hipDeviceReset must be called before exiting in order for profiling and
                // tracing tools such as Nsight and Visual Profiler to show complete traces.
                cudaStatus = hipDeviceReset();
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipDeviceReset failed!");
                    return 1;
                }
            }

            temps_cpu /= Ntest;
            temps_gpu *= 1000 / Ntest;

            float memoryUsed = 3.0 * arraySize * sizeof(int);
            float memoryThroughput = memoryUsed / temps_gpu / 1e+3; //Divide by 1 000 000 to have GB/s

            float numOperation = 1.0 * arraySize * k;
            // float memoryThroughput = memoryUsed / gpu_runtime_ms / 1e+6; //Divide by 1 000 000 to have GB/s
            float computationThroughput = numOperation / temps_gpu / 1e+3; // diviser par 1 000 car temps_gpu est en us
            float computeIntensity = computationThroughput / memoryThroughput;
            
            
            // std::cout << "Memory throughput : " << memoryThroughput << " GB/s " << std::endl;
            std::cout << "Computation throughput : " << computationThroughput << " GOPS/s " << std::endl;
            std::cout << "Compute intensity : " << computeIntensity << " OPS/Byte" << std::endl;

            /*
            std::cout << "mean: " << std::endl;
            std::cout << "Memory throughput : " << memoryThroughput << " GB/s " << std::endl;
            std::cout << "Computation throughput : " << computationThroughput << " GOPS/s " << std::endl;
            */

            // sauvegarde
            fichier << arraySize << "|";
            fichier << k << "|";
            fichier << temps_gpu << "|";
            fichier << memoryThroughput << "|";
            fichier << computationThroughput << "|";
            fichier << computeIntensity << std::endl;
        }
    }

    return 0;
}

bool checkCuda(int* out_cpu, int* out_gpu, int N) {
    bool res = true;
    for (int i = 0; i < N; i++) {
        if (out_cpu[i] != out_gpu[i]) {
            printf("ERROR : cpu : %d != gpu : %d \n", out_cpu[i], out_gpu[i]);
            res = false;
        }
    }
    return res;
}
